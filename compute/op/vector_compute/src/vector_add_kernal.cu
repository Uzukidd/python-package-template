#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void vector_add_kernel(
    const float* a,
    const float* b,
    float* c,
    int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

torch::Tensor vector_add_cuda(torch::Tensor a, torch::Tensor b) {
    TORCH_CHECK(a.sizes() == b.sizes(), "Input tensors must have same shape");

    auto c = torch::zeros_like(a);
    
    int threads = 256;
    int blocks = (a.numel() + threads - 1) / threads;
    
    vector_add_kernel<<<blocks, threads>>>(
        a.data_ptr<float>(),
        b.data_ptr<float>(),
        c.data_ptr<float>(),
        a.numel());
    
    return c;
}

